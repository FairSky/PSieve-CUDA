#include "hip/hip_runtime.h"
/* ex: set softtabstop=2 shiftwidth=2 expandtab: */
/* app.cu -- (C) Ken Brazier February 2010.

   Proth Prime Search sieve CUDA portion (for many K and many N).

   This program is free software; you can redistribute it and/or modify
   it under the terms of the GNU General Public License as published by
   the Free Software Foundation; either version 2 of the License, or
   (at your option) any later version.
*/

#include <stdio.h>
#include <stdint.h>
#include <string.h>
#include <inttypes.h>
#include <assert.h>
#include "main.h"
#include "util.h"
#include "app.h"
#include "appcu.h"
#define INLINE static inline
#define BITSATATIME 4
#define BITSMASK ((1<<BITSATATIME)-1)
// BLOCKSIZE should be a power of two for greatest efficiency.
#define BLOCKSIZE 128
#if(BITSATATIME == 3)
  #define SHIFT_CAST unsigned int
#elif(BITSATATIME == 4)
  #define SHIFT_CAST uint64_t
#else
  #error "Invalid BITSATATIME."
#endif
// Extern vars in appcu.h:
unsigned int ld_nstep;

// Device constants
__constant__ unsigned int d_bitsatatime;
__constant__ unsigned int d_len;//=(1<<bitsatatime); 
__constant__ unsigned int d_halflen;//=(1<<bitsatatime)/2; 
__constant__ uint64_t d_kmax;
__constant__ unsigned int d_bpernstep;
__constant__ unsigned int d_nmin;
__constant__ unsigned int d_nmax;
__constant__ unsigned int d_nstep;
__constant__ unsigned int d_search_proth;
// Device arrays
uint64_t *d_P;
uint64_t *d_K;
uint64_t *d_bitsskip;
unsigned char *d_factor_found;

/* This function is called once before any threads are started.
 */
unsigned int cuda_app_init(int gpuno)
{
  unsigned int i;
  struct hipDeviceProp_t gpuprop;
  unsigned int ld_bitsatatime = 0;
  //unsigned int ld_halflen=(1<<bitsatatime)/2; 
  unsigned int ld_bitsmask;
  unsigned int ld_bpernstep;
  unsigned int cthread_count;

  // Find the GPU's properties.
  if(hipGetDeviceProperties(&gpuprop, gpuno) != hipSuccess) {
    fprintf(stderr, "GPU %d not compute-capable.\n", gpuno);
    return 0;
  }
  hipSetDevice(gpuno);
  fprintf(stderr, "Detected GPU %d: %s\n", gpuno, gpuprop.name);
  fprintf(stderr, "Detected compute capability: %d.%d\n", gpuprop.major, gpuprop.minor);
  fprintf(stderr, "Detected %d multiprocessors.\n", gpuprop.multiProcessorCount);
  fprintf(stderr, "Detected %lu bytes of device memory.\n", gpuprop.totalGlobalMem);

  // Use them to set cthread_count.
  // First, threads per multiprocessor, based on compute capability.
  cthread_count = (gpuprop.major == 1 && gpuprop.minor < 2)?384:768;
  cthread_count *= gpuprop.multiProcessorCount;

  if(gpuprop.totalGlobalMem < cthread_count*48) {
    fprintf(stderr, "Insufficient GPU memory: %u bytes.\n",  (unsigned int)(gpuprop.totalGlobalMem));
    return 0;
  }
  // Calculate ld_bitsatatime given memory constraints, and possibly nmin-nmax via nstep vs. 2^ld_bitsatatime
  // Things change if nmax-nmin < 1000000 or so, but for now let's go with a constant maximum of ld_bitsatatime<=13.
  i = gpuprop.totalGlobalMem/sizeof(uint64_t); // Total number of 64-bit numbers that can be stored.
#ifndef NDEBUG
  fprintf(stderr, "Available memory = %d bytes\n", (int)(gpuprop.totalGlobalMem));
#endif
  ld_bitsatatime = BITSATATIME;
  ld_bitsmask = BITSMASK+1;

  // Allocate device arrays:
  // TODO: fix this awkward construct.
  while(1) {
    // - d_bitsskip[] (Biggest array first.)
    // Not using hipMallocPitch because coalescing isn't possible in general.
    if(hipMalloc((void**)&d_bitsskip, ld_bitsmask*cthread_count*sizeof(uint64_t)) == hipSuccess) {
      // - P's
      if(hipMalloc((void**)&d_P, cthread_count*sizeof(uint64_t)) == hipSuccess) {
        // - K's
        if(hipMalloc((void**)&d_K, cthread_count*sizeof(uint64_t)) == hipSuccess) {
          // - d_factor_found[]
          if(hipMalloc((void**)&d_factor_found, cthread_count*sizeof(unsigned char)) == hipSuccess) {
#ifndef NDEBUG
            fprintf(stderr, "Allocation successful!\n");
            fprintf(stderr, "ld_bitsatatime = %u\n", ld_bitsatatime);
#endif
            break;  // Allocation successful!
          }
          hipFree(d_K);
        }
        hipFree(d_P);
      }
      hipFree(d_bitsskip);
    }
    fprintf(stderr, "Insufficient available memory on GPU %d.\n", gpuno);
    return 0;
  }

  ld_bitsmask--; // Finalize bitsmask

  // Calculate the values that fit the given ld_bitsatatime.
  // ld_nstep was previously calculated in app_init.
  if (ld_nstep > ld_bitsatatime) {
    ld_bpernstep = ld_nstep/ld_bitsatatime;
    ld_nstep = ld_bpernstep*ld_bitsatatime;
  }
  if (ld_nstep > (nmax-nmin+1))
    ld_nstep = (nmax-nmin+1);

  // Set the constants.
  hipMemcpyToSymbol(HIP_SYMBOL(d_bitsatatime), &ld_bitsatatime, sizeof(ld_bitsatatime));

  // The following would be "pitch" if using hipMallocPitch above.
  i = ld_bitsmask+1;
  hipMemcpyToSymbol(HIP_SYMBOL(d_len), &i, sizeof(i));//=(1<<bitsatatime); 
  // But the following would not.
  i >>= 1;
  hipMemcpyToSymbol(HIP_SYMBOL(d_halflen), &i, sizeof(i));//=(1<<bitsatatime)/2; 
  hipMemcpyToSymbol(HIP_SYMBOL(d_kmax), &kmax, sizeof(kmax));
  hipMemcpyToSymbol(HIP_SYMBOL(d_bpernstep), &ld_bpernstep, sizeof(ld_bpernstep));
  hipMemcpyToSymbol(HIP_SYMBOL(d_nmin), &nmin, sizeof(nmin));
  hipMemcpyToSymbol(HIP_SYMBOL(d_nmax), &nmax, sizeof(nmax));
  hipMemcpyToSymbol(HIP_SYMBOL(d_nstep), &ld_nstep, sizeof(ld_nstep));
  hipMemcpyToSymbol(HIP_SYMBOL(d_search_proth), &search_proth, sizeof(search_proth));

  return cthread_count;
}



// Set up the lookup tables for all P's.
// TODO: Fix the variable names here to be more descriptive.
__global__ void d_setup_ps(const uint64_t *P, uint64_t *bitsskip) {
  unsigned int n, i;
  uint64_t *bs0 = &bitsskip[blockIdx.x * BLOCKSIZE*d_len + threadIdx.x];
  uint64_t my_P;
  uint64_t kpos;
  SHIFT_CAST mul_shift = 0, off_shift = 0;
  unsigned int my_factor_found = 0;
  // Initialize bitsskip array.
  my_P = P[blockIdx.x * BLOCKSIZE + threadIdx.x];

  // Initialize the first two entries.
  bs0[BLOCKSIZE*d_halflen] = (my_P+1)/2;	// Needed first.
  // bs0[0] will be ignored; it's just 0.

  // Fill in the intervening spaces, two numbers at a time.
  for(i=d_halflen; i > 1; i >>= 1) {
    for(n=i/2; n < d_halflen; n+=i) {
      kpos = bs0[BLOCKSIZE*2*n];
      my_factor_found = ((unsigned int)kpos)&1;
      //printf("Filling n=%d from bs0=%lu\n", n, kpos);
      bs0[BLOCKSIZE*n] = (kpos+((my_factor_found)?my_P:(uint64_t)0))/2;
      //printf("Filling n=%d\n", n+d_halflen);
      bs0[BLOCKSIZE*(n+d_halflen)] = (kpos+1+((my_factor_found)?(uint64_t)0:my_P))/2;
    }
  }
  // Now convert the entries to multiples of 1/8*P, plus some small constant.
  my_P >>= BITSATATIME;
  // Might as well skip 0, which is always 0.
  for(i=1; i < d_len; i++) {
    kpos = bs0[i*BLOCKSIZE];
    n = (unsigned int)(kpos/my_P);
    mul_shift |= ((SHIFT_CAST)n) << (BITSATATIME*i);
    off_shift |= (kpos-(((SHIFT_CAST)n)*my_P)) << (BITSATATIME*i);
  }

#if(BITSATATIME == 3)
  bs0[0] = (((uint64_t)off_shift)<<32) + mul_shift;
#elif(BITSATATIME == 4)
  bs0[0] = mul_shift;
  bs0[BLOCKSIZE] = off_shift;
#else
  #error "Invalid BITSATATIME."
#endif
}

// Check all N's.
__global__ void d_check_ns(const uint64_t *P, const uint64_t *K, unsigned char *factor_found_arr, uint64_t *bitsskip) {
  unsigned int n = d_nmin; // = nmin;
  unsigned int i = blockIdx.x * BLOCKSIZE + threadIdx.x;
  uint64_t k0;
  uint64_t kpos;
  unsigned char my_factor_found = 0;
  uint64_t my_P;
  unsigned int shift;
#ifndef NDEBUG
  uint64_t *bs0 = &bitsskip[blockIdx.x * BLOCKSIZE*d_len + threadIdx.x];
#endif
  SHIFT_CAST mul_shift, off_shift;

  //factor_found_arr[i] = 0;
  i = blockIdx.x * BLOCKSIZE*d_len + threadIdx.x;
#if(BITSATATIME == 3)
  my_P = bitsskip[i];
  mul_shift = (unsigned int)my_P;
  off_shift = (unsigned int)(my_P >> 32);
#elif(BITSATATIME == 4)
  mul_shift = bitsskip[i];
  off_shift = bitsskip[i+BLOCKSIZE];
#else
  #error "Invalid BITSATATIME."
#endif
  i = blockIdx.x * BLOCKSIZE + threadIdx.x;
  k0 = K[i];
  my_P = P[i];
  
  if(d_search_proth) k0 = my_P-k0;
  my_P >>= BITSATATIME;
  my_factor_found = 0;
  do { // Remaining steps are all of equal size nstep
    kpos = k0;
    i = __ffsll(kpos)-1;

    kpos >>= i;
    if (kpos <= d_kmax) {
#ifndef NDEBUG
      fprintf(stderr, "%u | %u*2^%u+1 (P[%d])\n", (unsigned int)my_P, (unsigned int)kpos, n+i, blockIdx.x * BLOCKSIZE + threadIdx.x);
#endif
      // Just flag this if kpos <= d_kmax.
      my_factor_found = 1;
    }

    for(i=0; i < d_bpernstep; i++) {
      shift=BITSATATIME*(((unsigned int)k0)&BITSMASK);
#ifndef NDEBUG
      if(shift > BITSATATIME && (bs0[((unsigned int)k0 & BITSMASK)*BLOCKSIZE] != ((((unsigned int)(mul_shift>>shift))&BITSMASK)*my_P + (((unsigned int)(off_shift>>shift))&BITSMASK)))) {
        fprintf(stderr, "Array lookup[%d], %lu != register lookup %lu\n", (unsigned int)k0 & BITSMASK, bs0[((unsigned int)k0 & BITSMASK)*BLOCKSIZE], ((((unsigned int)(mul_shift>>shift))&BITSMASK)*my_P + (((unsigned int)(off_shift>>shift))&BITSMASK)));
      }
      assert((shift == 0 && ((((unsigned int)(mul_shift>>shift))&BITSMASK)*my_P + (((unsigned int)(off_shift>>shift))&BITSMASK) == 0)) || shift == BITSATATIME || (bs0[((unsigned int)k0 & BITSMASK)*BLOCKSIZE] == ((((unsigned int)(mul_shift>>shift))&BITSMASK)*my_P + (((unsigned int)(off_shift>>shift))&BITSMASK))));
#endif
      k0 = (k0 >> BITSATATIME) + (((unsigned int)(mul_shift>>shift))&BITSMASK)*my_P + (((unsigned int)(off_shift>>shift))&BITSMASK);
    }
    n += d_nstep;
  } while (n < d_nmax);
  factor_found_arr[blockIdx.x * BLOCKSIZE + threadIdx.x] = my_factor_found;
}

// Pass the arguments to the CUDA device, run the code, and get the results.
void check_ns(const uint64_t *P, const uint64_t *K, unsigned char *factor_found, unsigned int cthread_count) {
  hipError_t res;
#ifndef NDEBUG
  fprintf(stderr, "In check_ns...\n");
#endif
  // Pass P.
  res = hipMemcpy(d_P, P, cthread_count*sizeof(uint64_t), hipMemcpyHostToDevice);
  if(res != hipSuccess) {
    if(res == hipErrorInvalidValue) fprintf(stderr, "Memcpy error: Invalid value!\n");
    if(res == hipErrorInvalidDevicePointer) fprintf(stderr, "Memcpy error: Invalid device pointer!\n");
    if(res == hipErrorInvalidMemcpyDirection) fprintf(stderr, "Memcpy error: Invalid memcpy direction!\n");
    exit(1);
  }
#ifndef NDEBUG
  fprintf(stderr, "Memcpy successful...\n");
#endif
  d_setup_ps<<<cthread_count/BLOCKSIZE,BLOCKSIZE>>>(d_P, d_bitsskip);
#ifndef NDEBUG
  fprintf(stderr, "Setup successful...\n");
#endif
  // Pass K.
  if(hipMemcpy(d_K, K, cthread_count*sizeof(uint64_t), hipMemcpyHostToDevice) != hipSuccess) {
    fprintf(stderr, "Memcpy2 error!\n");
    exit(1);
  }
#ifndef NDEBUG
  fprintf(stderr, "Memcpy2 successful...\n");
#endif
  d_check_ns<<<cthread_count/128,128>>>(d_P, d_K, d_factor_found, d_bitsskip);
#ifndef NDEBUG
  fprintf(stderr, "Main kernel successful...\n");
#endif
  // Get d_factor_found, into the thread'th factor_found array.
  hipMemcpy(factor_found, d_factor_found, cthread_count*sizeof(unsigned char), hipMemcpyDeviceToHost);
#ifndef NDEBUG
  fprintf(stderr, "Retrieve successful...\n");
#endif
}

void cuda_finalize(void) {
  hipFree(d_bitsskip);
  hipFree(d_K);
  hipFree(d_P);
  hipFree(d_factor_found);
}
