#include "hip/hip_runtime.h"
/* ex: set softtabstop=2 shiftwidth=2 expandtab: */
/* app.cu -- (C) Ken Brazier February 2010.

   Proth Prime Search sieve CUDA portion (for many K and many N).

   This program is free software; you can redistribute it and/or modify
   it under the terms of the GNU General Public License as published by
   the Free Software Foundation; either version 2 of the License, or
   (at your option) any later version.
*/

#include <stdio.h>
#include <stdint.h>
#include <string.h>
#include <inttypes.h>
#include <hip/hip_runtime.h>
//#include <assert.h>
#include "main.h"
#include "putil.h"
#include "app.h"
#include "appcu.h"

#define INLINE static inline
/*
#ifndef BITSATATIME
#define BITSATATIME 4
#endif
#define BITSMASK ((1<<BITSATATIME)-1)*/
/*
#if(BITSATATIME == 3)
  #define SHIFT_CAST unsigned int
#elif(BITSATATIME == 4)
  #define SHIFT_CAST uint64_t
#else
  #error "Invalid BITSATATIME."
#endif
*/
// Extern vars in appcu.h:
unsigned int ld_nstep;
int ld_bbits;
uint64_t ld_r0;

// Device constants
//__constant__ unsigned int d_bitsatatime;
//__constant__ unsigned int d_len;//=(1<<bitsatatime); 
//__constant__ unsigned int d_halflen;//=(1<<bitsatatime)/2; 
__constant__ uint64_t d_kmax;
__constant__ uint64_t d_kmin;
//__constant__ unsigned int d_bpernstep;
__constant__ unsigned int d_nmin;
__constant__ unsigned int d_nmax;
__constant__ unsigned int d_nstep;
__constant__ unsigned int d_kernel_nstep;
__constant__ unsigned int d_search_proth;

__constant__ int d_bbits;
__constant__ unsigned int d_mont_nstep;
__constant__ uint64_t d_r0;
// Device arrays
uint64_t *d_P;
uint64_t *d_Ps, *d_K;
//unsigned int *d_N;
unsigned char *d_factor_found;

// Timing variables:
//const int setup_ps_overlap = 5000;
//const int check_ns_overlap = 50000;

static unsigned int ld_kernel_nstep;

// find the log base 2 of a number.  Need not be fast; only done once.
int lg2(uint64_t v) {
	int r = 0; // r will be lg(v)

	while (v >>= 1) // unroll for more speed...
	{
		r++;
	}
	return r;
}
#ifndef _DEVICEEMU
bool SetCUDABlockingSync(int device) {
    hipDevice_t  hcuDevice;
    hipCtx_t hcuContext;

    hipError_t status = hipInit(0);
    if(status != hipSuccess)
       return false;

    status = hipDeviceGet( &hcuDevice, device);
    if(status != hipSuccess)
       return false;

    status = hipCtxCreate( &hcuContext, 0x4, hcuDevice );
    if(status != hipSuccess)
       return false;

    return true;
}
#endif
/* This function is called once before any threads are started.
 */
unsigned int cuda_app_init(int gpuno)
{
  unsigned int i;
  struct hipDeviceProp_t gpuprop;
  //unsigned int ld_bitsatatime = 0;
  //unsigned int ld_halflen=(1<<bitsatatime)/2; 
  //unsigned int ld_bitsmask;
  //unsigned int ld_bpernstep;
  unsigned int cthread_count;

#ifndef _DEVICEEMU
  SetCUDABlockingSync(gpuno);
#endif
  // Find the GPU's properties.
  if(hipGetDeviceProperties(&gpuprop, gpuno) != hipSuccess) {
    fprintf(stderr, "%sGPU %d not compute-capable.\n", bmprefix(), gpuno);
    return 0;
  }
  /* Assume N >= 2^32. */
  if(pmin <= ((uint64_t)1)<<32) {
    bmsg("Error: PMin is too small, <= 2^32!\n");
    bexit(1);
  }
  hipSetDevice(gpuno);
  fprintf(stderr, "%sDetected GPU %d: %s\n", bmprefix(), gpuno, gpuprop.name);
  fprintf(stderr, "%sDetected compute capability: %d.%d\n", bmprefix(), gpuprop.major, gpuprop.minor);
  fprintf(stderr, "%sDetected %d multiprocessors.\n", bmprefix(), gpuprop.multiProcessorCount);
  //fprintf(stderr, "%sDetected %lu bytes of device memory.\n", bmprefix(), gpuprop.totalGlobalMem);

  // Use them to set cthread_count.
  // First, threads per multiprocessor, based on compute capability.
  cthread_count = (gpuprop.major == 1 && gpuprop.minor < 2)?384:768;
  if(gpuprop.major == 2) cthread_count = 1024;
  cthread_count *= gpuprop.multiProcessorCount;

  if(gpuprop.totalGlobalMem < cthread_count*5) {
    fprintf(stderr, "%sInsufficient GPU memory: %u bytes.\n", bmprefix(), (unsigned int)(gpuprop.totalGlobalMem));
    return 0;
  }
  // Calculate ld_bitsatatime given memory constraints, and possibly nmin-nmax via nstep vs. 2^ld_bitsatatime
  // Things change if nmax-nmin < 1000000 or so, but for now let's go with a constant maximum of ld_bitsatatime<=13.
  i = gpuprop.totalGlobalMem/sizeof(uint64_t); // Total number of 64-bit numbers that can be stored.
  //ld_bitsatatime = BITSATATIME;
  //ld_bitsmask = BITSMASK+1;

  // Allocate device arrays:
  // TODO: fix this awkward construct.
  while(1) {
    // - d_bitsskip[] (Biggest array first.)
    //if(hipMalloc((void**)&d_bitsskip, ld_bitsmask*cthread_count*sizeof(uint64_t)) == hipSuccess) {
      // - P's
      if(hipMalloc((void**)&d_P, cthread_count*sizeof(uint64_t)) == hipSuccess) {
        // - Ps's
        if(hipMalloc((void**)&d_Ps, cthread_count*sizeof(uint64_t)) == hipSuccess) {
          // - K's
          if(hipMalloc((void**)&d_K, cthread_count*sizeof(uint64_t)) == hipSuccess) {
            // - N's
            //if(hipMalloc((void**)&d_N, cthread_count*sizeof(unsigned int)) == hipSuccess) {
              // - d_factor_found[]
              if(hipMalloc((void**)&d_factor_found, cthread_count*sizeof(unsigned char)) == hipSuccess) {
#ifndef NDEBUG
                fprintf(stderr, "Allocation successful!\n");
                fprintf(stderr, "ld_bitsatatime = %u\n", ld_bitsatatime);
#endif
                break;  // Allocation successful!
              }
			  //hipFree(d_N);
			//}
            hipFree(d_K);
          }
          hipFree(d_Ps);
        }
        hipFree(d_P);
      }
      //hipFree(d_bitsskip);
    //}
    fprintf(stderr, "%sInsufficient available memory on GPU %d.\n", bmprefix(), gpuno);
    return 0;
  }

  //ld_bitsmask--; // Finalize bitsmask

  if (ld_nstep > (nmax-nmin+1))
    ld_nstep = (nmax-nmin+1);

  //assert((1ul << (64-nstep)) < pmin);
  if((((uint64_t)1) << (64-ld_nstep)) > pmin) {
    bmsg("Error: pmin is not large enough (or nmax is close to nmin).\n");
    bexit(1);
  }
  // Set the constants.
  //hipMemcpyToSymbol(HIP_SYMBOL(d_bitsatatime), &ld_bitsatatime, sizeof(ld_bitsatatime));

  // Prepare constants:
  ld_bbits = lg2(nmin);
  //assert(d_r0 <= 32);
  if(ld_bbits < 6) {
    fprintf(stderr, "%sError: nmin too small at %d (must be at least 64).\n", bmprefix(), nmin);
    bexit(1);
  }
  // r = 2^-i * 2^64 (mod N), something that can be done in a uint64_t!
  // If i is large (and it should be at least >= 32), there's a very good chance no mod is needed!
  ld_r0 = ((uint64_t)1) << (64-(nmin >> (ld_bbits-5)));

  ld_bbits = ld_bbits-6;
  hipMemcpyToSymbol(HIP_SYMBOL(d_bbits), &ld_bbits, sizeof(ld_bbits));
  // d_mont_nstep is the montgomerized version of nstep.
  i = 64-ld_nstep;
  hipMemcpyToSymbol(HIP_SYMBOL(d_mont_nstep), &i, sizeof(i));
  hipMemcpyToSymbol(HIP_SYMBOL(d_r0), &ld_r0, sizeof(ld_r0));
  // N's to search each time a kernel is run:
  ld_kernel_nstep = ITERATIONS_PER_KERNEL * ld_nstep;
  // Adjust for differing block sizes.
  ld_kernel_nstep *= 384;
  ld_kernel_nstep /= (cthread_count/gpuprop.multiProcessorCount);
  hipMemcpyToSymbol(HIP_SYMBOL(d_kernel_nstep), &ld_kernel_nstep, sizeof(ld_kernel_nstep));
  hipMemcpyToSymbol(HIP_SYMBOL(d_kmax), &kmax, sizeof(kmax));
  hipMemcpyToSymbol(HIP_SYMBOL(d_kmin), &kmin, sizeof(kmin));
  hipMemcpyToSymbol(HIP_SYMBOL(d_nmin), &nmin, sizeof(nmin));
  hipMemcpyToSymbol(HIP_SYMBOL(d_nmax), &nmax, sizeof(nmax));
  hipMemcpyToSymbol(HIP_SYMBOL(d_nstep), &ld_nstep, sizeof(ld_nstep));
  i = (search_proth == 1)?1:0;	// search_proth is 1 or -1, not 0.
  hipMemcpyToSymbol(HIP_SYMBOL(d_search_proth), &i, sizeof(i));


  return cthread_count;
}


/*** Kernel Helpers ***/
// Special thanks to Alex Kruppa for introducing me to Montgomery REDC math!
/* Compute a^{-1} (mod 2^(32 or 64)), according to machine's word size */

// Inside this #ifdef is code used only to check the faster code below it.
#ifndef NDEBUG
#ifdef __x86_64__
#define DEBUG64
/* Reduce a*2^64+b modulo m. Requires a < m, or the quotient (which we don't care about but the chip does) will overflow. */ 
__device__ uint64_t
longmod (uint64_t a, uint64_t b, const uint64_t m)
{
  //ASSERT (a < m);
  __asm__
  ( "divq %2"
    : "+d" (a), /* Put "a" in %rdx, will also get result of mod */
      "+a" (b)  /* Put "b" in %rax, will also be written to 
                   (quotient, which we don't need) */
    : "rm" (m)  /* Modulus can be in a register or memory location */
    : "cc"      /* Flags are clobbered */
  );
  return a;
}

//Now, for modulus!  From http://www.loria.fr/~kruppaal/factorcyc.20090612.c
/* Multiply a and b, and reduce the product modulo m. Remainder is
   returned */
// Be careful if a and b are >> m, as the quotient overflow from longmod could happen here too. :(
// But if either one is <= m, it's fine.
//#ifdef DEBUG
__device__ uint64_t mulmod (uint64_t a, const uint64_t b, const uint64_t m)
{
  uint64_t q, r, t1, t2;
  __asm__
  ( "mulq %3\n\t"
    : "=a" (t1), "=d" (t2)
    : "0" (a), "rm" (b)
    : "cc");
  __asm__
  ( "divq %4"
    : "=a" (q), "=d" (r)
    : "0" (t1), "1" (t2), "rm" (m)
    : "cc"
  );
  return r;
} 


/* Compute REDC(a*b) for modulus N. We need N*Ns == -1 (mod 2^64) */
__device__ uint64_t
asm_mulmod_REDC (const uint64_t a, const uint64_t b, 
             const uint64_t N, const uint64_t Ns)
{
	uint64_t r;

	// Akruppa's way, Compute T=a*b; m = (T*Ns)%2^64; T += m*N; if (T>N) T-= N;
	__asm__
		( "mulq %[b]\n\t"           // rdx:rax = T 			Cycles 1-7
		  "movq %%rdx,%%rcx\n\t"	// rcx = Th			Cycle  8
		  "imulq %[Ns], %%rax\n\t"  // rax = (T*Ns) mod 2^64 = m 	Cycles 8-12 
		  "cmpq $1,%%rax \n\t"      // if rax != 0, increase rcx 	Cycle 13
		  "sbbq $-1,%%rcx\n\t"	//				Cycle 14-15
		  "mulq %[N]\n\t"           // rdx:rax = m * N 		Cycle 13?-19?
		  "lea (%%rcx,%%rdx,1), %[r]\n\t" // compute (rdx + rcx) mod N  C 20 
		  "subq %[N], %%rcx\n\t"	//				Cycle 20/19?
		  "addq %%rdx, %%rcx\n\t"	//				Cycle 21/20?
		  "cmovcq %%rcx, %[r]\n\t"	//				Cycle 22/21?
		  : [r] "=r" (r)
		  : "%a" (a), [b] "rm" (b), [N] "rm" (N), [Ns] "rm" (Ns)
		  : "cc", "%rcx", "%rdx"
		);

#ifdef DEBUG64
	if (longmod (r, 0, N) != mulmod(a, b, N))
	{
		fprintf (stderr, "%sError, asm mulredc(%lu,%lu,%lu) = %lu\n", bmprefix(), a, b, N, r);
		abort();
	}
#endif

   return r;
}
#endif
#endif
__device__ uint64_t
invmod2pow_ul (const uint64_t n)
{
  uint64_t r;
  //unsigned int ir;
  const unsigned int in = (unsigned int)n;

  //ASSERT (n % 2UL != 0UL);
  
  // Suggestion from PLM: initing the inverse to (3*n) XOR 2 gives the
  // correct inverse modulo 32, then 3 (for 32 bit) or 4 (for 64 bit) 
  // Newton iterations are enough.
  r = (n+n+n) ^ ((uint64_t)2);
  // Newton iteration
  r += r - (unsigned int) r * (unsigned int) r * in;
  r += r - (unsigned int) r * (unsigned int) r * in;
  r += r - (unsigned int) r * (unsigned int) r * in;
  r += r - r * r * n;

  return r;
}

__device__ uint64_t mulmod_REDC (const uint64_t a, const uint64_t b, 
             const uint64_t N, const uint64_t Ns)
{
  uint64_t rax, rcx;

  // Akruppa's way, Compute T=a*b; m = (T*Ns)%2^64; T += m*N; if (T>N) T-= N;
  //( "mulq %[b]\n\t"           // rdx:rax = T 			Cycles 1-7
  rax = a*b;
  rcx = __umul64hi(a,b);
  //"movq %%rdx,%%rcx\n\t"	// rcx = Th			Cycle  8
  //rcx = rdx;
  //"imulq %[Ns], %%rax\n\t"  // rax = (T*Ns) mod 2^64 = m 	Cycles 8-12 
  rax *= Ns;
  //"cmpq $1,%%rax \n\t"      // if rax != 0, increase rcx 	Cycle 13
  //"sbbq $-1,%%rcx\n\t"	//				Cycle 14-15
  rcx += (rax!=0)?1:0;
  //"mulq %[N]\n\t"           // rdx:rax = m * N 		Cycle 13?-19?
  rax = __umul64hi(rax, N);
  //"lea (%%rcx,%%rdx,1), %[r]\n\t" // compute (rdx + rcx) mod N  C 20 
  rax += rcx;
  rcx = rax - N;
  rax = (rax>N)?rcx:rax;

#ifdef DEBUG64
  if (longmod (rax, 0, N) != mulmod(a, b, N))
  {
    fprintf (stderr, "%sError, mulredc(%lu,%lu,%lu) = %lu\n", bmprefix(), a, b, N, rax);
    bexit(1);
  }
#endif

  return rax;
}

// mulmod_REDC(1, 1, N, Ns)
// But note that mulmod_REDC(a, 1, N, Ns) == mulmod_REDC(1, 1, N, Ns*a).
__device__ uint64_t onemod_REDC(const uint64_t N, uint64_t rax) {
  uint64_t rcx;

  // Akruppa's way, Compute T=a*b; m = (T*Ns)%2^64; T += m*N; if (T>N) T-= N;
  //rcx = 0;
  //"cmpq $1,%%rax \n\t"      // if rax != 0, increase rcx 	Cycle 13
  //"sbbq $-1,%%rcx\n\t"	//				Cycle 14-15
  rcx = (rax!=0)?1:0;
  //"mulq %[N]\n\t"           // rdx:rax = m * N 		Cycle 13?-19?
  rax = __umul64hi(rax, N) + rcx;
  //"lea (%%rcx,%%rdx,1), %[r]\n\t" // compute (rdx + rcx) mod N  C 20 
  rcx = rax - N;
  rax = (rax>N)?rcx:rax;

  return rax;
}

// Like mulmod_REDC(a, 1, N, Ns) == mulmod_REDC(1, 1, N, Ns*a).
__device__ uint64_t mod_REDC(const uint64_t a, const uint64_t N, const uint64_t Ns) {
#ifndef DEBUG64
  return onemod_REDC(N, Ns*a);
#else
  const uint64_t r = onemod_REDC(N, Ns*a);

  if (longmod (r, 0, N) != mulmod(a, 1, N)) {
    fprintf (stderr, "%sError, redc(%lu,%lu) = %lu\n", bmprefix(), a, N, r);
    bexit(1);
  }

  return r;
#endif
}

// Compute T=a<<s; m = (T*Ns)%2^64; T += m*N; if (T>N) T-= N;
// rax is passed in as a * Ns.
// rax's original value is destroyed, just to keep the register count down.
__device__ uint64_t shiftmod_REDC (const uint64_t a, 
             const uint64_t N, uint64_t &rax)
{
  uint64_t rcx;

  //( "mulq %[b]\n\t"           // rdx:rax = T 			Cycles 1-7
  rax <<= d_mont_nstep; // So this is a*Ns*(1<<s) == (a<<s)*Ns.
  rcx = a >> d_nstep;
  //"movq %%rdx,%%rcx\n\t"	// rcx = Th			Cycle  8
  //"imulq %[Ns], %%rax\n\t"  // rax = (T*Ns) mod 2^64 = m 	Cycles 8-12 
  //rax *= Ns;
  //"cmpq $1,%%rax \n\t"      // if rax != 0, increase rcx 	Cycle 13
  //"sbbq $-1,%%rcx\n\t"	//				Cycle 14-15
  rcx += (rax!=0)?1:0;
  //"mulq %[N]\n\t"           // rdx:rax = m * N 		Cycle 13?-19?
  rax = __umul64hi(rax, N) + rcx;
  //"lea (%%rcx,%%rdx,1), %[r]\n\t" // compute (rdx + rcx) mod N  C 20 
  rcx = rax - N;
  rax = (rax>N)?rcx:rax;

#ifdef DEBUG64
  if (longmod (rax, 0, N) != mulmod(a, ((uint64_t)1)<<d_mont_nstep, N))
  {
    fprintf (stderr, "%sError, shiftredc(%lu,%u,%lu) = %lu\n", bmprefix(), a, d_mont_nstep, N, rax);
    bexit(1);
  }
#endif

  return rax;
}

// A Left-to-Right version of the powmod.  Calcualtes 2^-(first 6 bits), then just keeps squaring and dividing by 2 when needed.
__device__ uint64_t
invpowmod_REDClr (const uint64_t N, const uint64_t Ns) {
  uint64_t r;
  int bbits = d_bbits;

  r = d_r0;

  // Now work through the other bits of nmin.
  for(; bbits >= 0; --bbits) {
    // Just keep squaring r.
    r = mulmod_REDC(r, r, N, Ns);
    // If there's a one bit here, multiply r by 2^-1 (aka divide it by 2 mod N).
    if(d_nmin & (1u << bbits)) {
      r += (r&1)?N:0;
      r >>= 1;
    }
  }

#ifdef DEBUG64
  //assert (mod_REDC (r, N, Ns) == invmod(powmod (d_nmin, N), N));
#endif

  // Convert back to standard.
  //r = mod_REDC (r, N, Ns);

  return r;
}

// Device-local function to iterate over some N's.
// To avoid register pressure, clobbers i, and changes all non-const arguments.
__device__ void d_check_some_ns(const uint64_t my_P, const uint64_t Ps, uint64_t &k0,
								unsigned int &n, unsigned char &my_factor_found, unsigned int &i) {
  uint64_t kpos, kPs;
  unsigned int l_nmax = n + d_kernel_nstep;
  if(l_nmax > d_nmax) l_nmax = d_nmax;

#ifdef _DEVICEEMU
  if(my_P == 42070000070587) printf("Started at n=%u, k=%u; running %u n's (GPU)\n", n, (unsigned int)k0, d_kernel_nstep);
#endif
  do { // Remaining steps are all of equal size nstep
    // Get K from the Montgomery form.
    // This is equivalent to mod_REDC(k, my_P, Ps), but the intermediate kPs value is kept for later.
    kPs = k0 * Ps;
    kpos = onemod_REDC(my_P, kPs);
    //i = __ffsll(kpos)-1;
    i = (unsigned int)kpos;
    if(i != 0) {
      i=(__float_as_int(__uint2float_rz(i & -i))>>23)-0x7f;
    } else {
      i = (unsigned int)(kpos>>32);
      i=63 - __clz (i & -i);
    }

    kpos >>= i;
    if (kpos <= d_kmax) {
#ifdef _DEVICEEMU
    //fprintf(stderr, "%s%u | %u*2^%u+1 (P[%d])\n", bmprefix(), (unsigned int)my_P, (unsigned int)kpos, n+i, blockIdx.x * BLOCKSIZE + threadIdx.x);
#endif
      // Just flag this if kpos <= d_kmax.
      if(kpos >= d_kmin) my_factor_found = 1;
    }

    // Proceed to the K for the next N.
    // kPs is destroyed, just to keep the register count down.
    k0 = shiftmod_REDC(k0, my_P, kPs);
    n += d_nstep;
  } while (n < l_nmax);
#ifdef _DEVICEEMU
  if(my_P == 42070000070587) printf("Stopped at n=%u, k=%u (GPU)\n", n, (unsigned int)k0);
#endif
}

// *** KERNELS ***

// Start checking N's.
__global__ void d_start_ns(const uint64_t *P, uint64_t *Ps, uint64_t *K, unsigned char *factor_found_arr) {
  unsigned int n = d_nmin; // = nmin;
  unsigned int i = blockIdx.x * BLOCKSIZE + threadIdx.x;
  uint64_t k0;
  //unsigned char my_factor_found = 0;
  uint64_t my_P, my_Ps;
  my_P = P[i];
  
  // Better get this done before the first mulmod.
  my_Ps = -invmod2pow_ul (my_P); /* Ns = -N^{-1} % 2^64 */
  
  // Calculate k0, in Montgomery form.
  k0 = invpowmod_REDClr(my_P, my_Ps);

  //if(my_P == 42070000070587) printf("%lu^-1 = %lu (GPU)\n", my_P, my_Ps);

  if(d_search_proth) k0 = my_P-k0;

  //my_factor_found = 0;
  //d_check_some_ns(my_P, my_Ps, k0, n, my_factor_found, i);

  i = blockIdx.x * BLOCKSIZE + threadIdx.x;
  factor_found_arr[i] = 0;
  if(n < d_nmax) {
    Ps[i] = my_Ps;
	K[i] = k0;
  }
}

// Continue checking N's.
__global__ void d_check_more_ns(const uint64_t *P, const uint64_t *Ps, uint64_t *K, unsigned int N, unsigned char *factor_found_arr) {
  unsigned int n = N;
  unsigned int i = blockIdx.x * BLOCKSIZE + threadIdx.x;
  uint64_t k0 = K[i];
  unsigned char my_factor_found = factor_found_arr[i];

  d_check_some_ns(P[i], Ps[i], k0, n, my_factor_found, i);

  i = blockIdx.x * BLOCKSIZE + threadIdx.x;
  factor_found_arr[i] = my_factor_found;
  if(n < d_nmax) {
	K[i] = k0;
  }
}

// *** Host Kernel-calling functions ***

// Pass the arguments to the CUDA device, run the code, and get the results.
void check_ns(const uint64_t *P, const unsigned int cthread_count) {
  unsigned int n;
  // timing variables:
  hipError_t res;
  // Pass P.
  res = hipMemcpy(d_P, P, cthread_count*sizeof(uint64_t), hipMemcpyHostToDevice);
  if(res != hipSuccess) {
    if(res == hipErrorInvalidValue) bmsg("Memcpy error: Invalid value!\n");
    if(res == hipErrorInvalidDevicePointer) bmsg("Memcpy error: Invalid device pointer!\n");
    if(res == hipErrorInvalidMemcpyDirection) bmsg("Memcpy error: Invalid memcpy direction!\n");
    bexit(1);
  }
#ifndef NDEBUG
  bmsg("Setup successful...\n");
#endif
  d_start_ns<<<cthread_count/128,128>>>(d_P, d_Ps, d_K, d_factor_found);
#ifndef NDEBUG
  bmsg("Main kernel successful...\n");
#endif
  // Continue checking until nmax is reached.
  for(n = nmin; n < nmax; n += ld_kernel_nstep) {
    d_check_more_ns<<<cthread_count/128,128>>>(d_P, d_Ps, d_K, n, d_factor_found);
  }
}

void get_factors_found(unsigned char *factor_found, const unsigned int cthread_count) {
  // Get d_factor_found, into the thread'th factor_found array.
  hipMemcpy(factor_found, d_factor_found, cthread_count*sizeof(unsigned char), hipMemcpyDeviceToHost);
#ifndef NDEBUG
  bmsg("Retrieve successful...\n");
#endif
}

void cuda_finalize(void) {
  //hipFree(d_bitsskip);
//  hipFree(d_N);
  hipFree(d_K);
  hipFree(d_Ps);
  hipFree(d_P);
  hipFree(d_factor_found);
}
